//Konstantinos Bourantas 
//AM: 23 6145
//bourantas@ceid.upatras.gr

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <string>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>
///////////////////////////////////////////////////////////////////////////////////////////////////////////

#define BLOCK_SIZE 16

using namespace std;

///////////////////////////////////////////////////////////////////////////////////////////////////////////
#define cudaCheckError() {                               \
                hipError_t e = hipGetLastError();                    \
                if (e != hipSuccess) {                                \
                        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
                               hipGetErrorString(e));                      \
                        exit(1);                                             \
                }                                                      \
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////

void calculateElapsedTime(int iterations, float time_elapsed){
       
  cout << "Average time elapsed: " << time_elapsed/iterations << " seconds" << endl;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////
//Print matrix A(rows_A, cols_A) storage in column-major format
void print_matrix(double const * const A, const int rows, const int cols) {
	int i, j;
	for (i = 0; i < rows; ++i) {
		for (j = 0; j < cols; ++j) {
			printf("%.2lf ", A[i * cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void multiplicationKernel(double const * const input_matrix, double * const output_matrix, const int rows_A, const int cols_A) {
	
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
                
	double tempSum = 0.0;
    
	if (row < cols_A && col < cols_A) 
	{
		for (int i = 0; i < rows_A; ++i) {
            // printf("multiplying A[%d]=%f * A[%d]=%f\n",i * cols_A + row, input_matrix[i * cols_A + row],i * cols_A + col,input_matrix[i * cols_A + col]);
			tempSum += input_matrix[i * cols_A + row] * input_matrix[i * cols_A + col];
		}
		output_matrix[row * cols_A + col] = tempSum;
	}
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

	//fixed srand for debugging
	srand(time(0));

	//select device
	hipSetDevice(0);
	cudaCheckError();

	if (argc < 4)
	{
		cout << "Usage: " << argv[0] << "[Rows]-[Cols]-[Iterations]" << endl;
		return 1;
	}

	//user input
	unsigned int rows_A, cols_A, rows_C, cols_C, rows_A_T, cols_A_T;

	rows_A = cols_A_T = atoi(argv[1]);
	cols_A = rows_A_T = rows_C = cols_C = atoi(argv[2]);
	int iterations = atoi(argv[3]);

	hipEvent_t start, stop;

	// Allocate host_A and host_C on host
	double *host_A = (double*) malloc(rows_A *cols_A* sizeof(double));
	double *host_C = (double*) malloc(rows_C *cols_C* sizeof(double));

	//fill array host_A with values
	double random_data = 1.0;
	for (int i = 0; i < rows_A * cols_A; i++)
	{
		host_A[i] = random_data;
		printf("host_A[%d]=%f",i,host_A[i]);
		random_data += 1.0;
	}

	print_matrix(host_A,rows_A,cols_A);

	// Allocate array device_A and deive_C on GPU memory
	double *device_A;
	hipMalloc(&device_A, rows_A *cols_A* sizeof(double));
	cudaCheckError();

	double *device_C;
	hipMalloc(&device_C, rows_C *cols_C* sizeof(double));
	cudaCheckError();

	//Copy array host_A to gpu thought device_A
	hipMemcpy(device_A, host_A, rows_A *cols_A* sizeof(double), hipMemcpyHostToDevice);
	cudaCheckError();

	//create cuda events for time measuring
	hipEventCreate(&start);
	cudaCheckError();
	hipEventCreate(&stop);
	cudaCheckError();

	float time_elapsed = 0;
	float tempTime;

	cout << "Executing operation for matrix A with " <<
		cols_A << " columns and " << rows_A << " rows." << endl;
	cout << "=======================================" << endl;

	//begin iterations
	for (int i = 0; i < iterations; i++)
	{

		tempTime = 0;

		unsigned int grid_rows = (cols_A + BLOCK_SIZE - 1) / BLOCK_SIZE;
		unsigned int grid_cols = (rows_A + BLOCK_SIZE - 1) / BLOCK_SIZE;
		dim3 dimGrid(grid_cols, grid_rows);
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

		// Create cublas handle
		hipblasHandle_t handle;
		cudaCheckError();
		hipblasCreate(&handle);
		cudaCheckError();
		hipEventRecord(start);
		cudaCheckError();

		//run gpu kernel
		multiplicationKernel <<<dimGrid, dimBlock>>> (device_A, device_C, rows_A, cols_A);
		cudaCheckError();

		hipEventRecord(stop);
		cudaCheckError();

		// Destroy the handle
		hipblasDestroy(handle);
		cudaCheckError();

		// Copy the resulted matrix device_C to host
		hipMemcpy(host_C, device_C, rows_C *cols_C* sizeof(double), hipMemcpyDeviceToHost);
		cudaCheckError();

		hipEventSynchronize(stop);
		cudaCheckError();

		// Compute the elapsed time between the two events start and stop
		hipEventElapsedTime(&tempTime, start, stop);
		cudaCheckError();

		cout << "Iteration " << i + 1 << ": " << tempTime << " seconds" << endl;
		time_elapsed += tempTime;
	}

	print_matrix(host_C, rows_C, cols_C);

	hipEventDestroy(start);
	cudaCheckError();

	hipEventDestroy(stop);
	cudaCheckError();

	cout << "=======================================" << endl;

	//calculate average elapsed time
	calculateElapsedTime(iterations, time_elapsed);

	//Free memory on host and device
	hipFree(device_A);
	cudaCheckError();
	free(host_A);

	hipFree(device_C);
	cudaCheckError();
	free(host_C);

	return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
