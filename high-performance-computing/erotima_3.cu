//Konstantinos Bourantas 
//AM: 23 6145
//bourantas@ceid.upatras.gr


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <string>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>
///////////////////////////////////////////////////////////////////////////////////////////////////////////


#define TILE_SIZE 16
#define BLOCK_SIZE 16

using namespace std;

///////////////////////////////////////////////////////////////////////////////////////////////////////////
#define cudaCheckError() {                               \
                hipError_t e = hipGetLastError();                    \
                if (e != hipSuccess) {                                \
                        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
                               hipGetErrorString(e));                      \
                        exit(1);                                             \
                }                                                      \
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////

void calculateElapsedTime(int iterations, float time_elapsed){
       
  cout << "Average time elapsed: " << time_elapsed/iterations << " seconds" << endl;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////
//Print matrix A(rows_A, cols_A) storage in column-major format
void print_matrix(double
	const *const A, const int rows, const int cols)
{
	int i, j;
	for (i = 0; i < rows; ++i)
	{
		for (j = 0; j < cols; ++j)
		{
			printf("%.2lf ", A[i *cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}



///////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void matrixMultKernel(double* A,  double* C, int rows_A, int cols_A ,int rows_C, int cols_C)
{
    float temp_sum = 0;

    int Row = blockIdx.y*TILE_SIZE + threadIdx.y;
    int Col = blockIdx.x*TILE_SIZE + threadIdx.x;

    __shared__ float A_shared[TILE_SIZE][TILE_SIZE+1];
    __shared__ float A_T_shared[TILE_SIZE][TILE_SIZE+1];

    for (int i = 0; i < (TILE_SIZE +cols_A - 1)/TILE_SIZE; i++) {
    
         if (i*TILE_SIZE + threadIdx.y <cols_A && Col < rows_A)
             A_T_shared[threadIdx.y][threadIdx.x] = A[(i*TILE_SIZE+threadIdx.y)+Col*rows_A];
         else
             A_T_shared[threadIdx.y][threadIdx.x] = 0.0;

         if (i*TILE_SIZE + threadIdx.x <cols_A && Row < rows_A)
             A_shared[threadIdx.y][threadIdx.x] = A[Row*cols_A+ i*TILE_SIZE + threadIdx.x];
         else
             A_shared[threadIdx.y][threadIdx.x] = 0.0;

         
         __syncthreads();

         for (int n = 0; n < TILE_SIZE; ++n)
             temp_sum += A_T_shared[threadIdx.y][n] * A_shared[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < rows_C && Col < cols_C){

	C[Col*rows_C+Row] = temp_sum;

}
		

}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

	//fixed srand for debugging
	srand(time(0));

	//select device
	hipSetDevice(0);
	cudaCheckError();

	if (argc < 4)
	{
		cout << "Usage: " << argv[0] << "[Rows]-[Cols]-[Iterations]" << endl;
		return 1;
	}

	//user input
	unsigned int rows_A, cols_A, rows_C, cols_C, rows_A_T, cols_A_T;

	rows_A = cols_A_T = atoi(argv[1]);
	cols_A = rows_A_T = rows_C = cols_C = atoi(argv[2]);
	int iterations = atoi(argv[3]);

	hipEvent_t start, stop;

	// Allocate host_A and host_C on host
	double *host_A = (double*) malloc(rows_A *cols_A* sizeof(double));
	double *host_C = (double*) malloc(rows_C *cols_C* sizeof(double));

	//fill array host_A with values
	double random_data = 1.0;
  for (int i = 0; i < rows_A * cols_A; i++)
	{
		host_A[i] = random_data;
		random_data += 1.0;
	}
 
	print_matrix(host_A, rows_A, cols_A);

	// Allocate array device_A and deive_C on GPU memory
	double *device_A;
	hipMalloc(&device_A, rows_A *cols_A* sizeof(double));
	cudaCheckError();
 

	double *device_C;
	hipMalloc(&device_C, rows_C *cols_C* sizeof(double));
	cudaCheckError();

	//Copy array host_A to gpu thought device_A
	hipMemcpy(device_A, host_A, rows_A *cols_A* sizeof(double), hipMemcpyHostToDevice);
	cudaCheckError();
 	

	//create cuda events for time measuring
	hipEventCreate(&start);
	cudaCheckError();
	hipEventCreate(&stop);
	cudaCheckError();
 	

	float time_elapsed = 0;
	float tempTime;

	cout << "Executing operation for matrix A with " <<
		cols_A << " columns and " << rows_A << " rows." << endl;
	cout << "=======================================" << endl;

	//begin iterations
	for (int i = 0; i < iterations; i++)
	{

		tempTime = 0;

		// Create cublas handle
		hipblasHandle_t handle;
		cudaCheckError();
		hipblasCreate(&handle);
		cudaCheckError();
		hipEventRecord(start);
		cudaCheckError();

		
		dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
		dim3 dimGrid;

		dimGrid.x = (cols_C + dimBlock.x - 1)/dimBlock.x;
		dimGrid.y = (rows_C+ dimBlock.y - 1)/dimBlock.y;



		// Launch kernel////////////////////////////////////////////////////

		matrixMultKernel <<<dimGrid, dimBlock>>>(device_A, device_C, rows_A,cols_A, rows_C, cols_C);
		cudaCheckError();
		
		////////////////////////////////////////////////////////////////////

		hipEventRecord(stop);
		cudaCheckError();

		// Destroy the handle
		hipblasDestroy(handle);
		cudaCheckError();

		// Copy the resulted matrix device_C to host
		hipMemcpy(host_C, device_C, rows_C *cols_C* sizeof(double), hipMemcpyDeviceToHost);
		cudaCheckError();

		hipEventSynchronize(stop);
		cudaCheckError();

		// Compute the elapsed time between the two events start and stop
		hipEventElapsedTime(&tempTime, start, stop);
		cudaCheckError();

		cout << "Iteration " << i + 1 << ": " << tempTime << " seconds" << endl;
		time_elapsed += tempTime;
	}

	print_matrix(host_C, rows_C, cols_C);

	hipEventDestroy(start);
	cudaCheckError();

	hipEventDestroy(stop);
	cudaCheckError();

	cout << "=======================================" << endl;

	//calculate average elapsed time
	calculateElapsedTime(iterations, time_elapsed);

	//Free memory on host and device
	hipFree(device_A);
	cudaCheckError();
	free(host_A);

	hipFree(device_C);
	cudaCheckError();
	free(host_C);

	return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
