//Konstantinos Bourantas 
//AM: 23 6145
//bourantas@ceid.upatras.gr


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <string>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>
#include <fstream>

using namespace std;
/////////////////////////////////////////////////////////////////////////////////////////////////////

#define cudaCheckError() {                               \
        hipError_t e = hipGetLastError();                    \
        if (e != hipSuccess) {                                \
                printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
                       hipGetErrorString(e));                      \
                exit(1);                                             \
        }                                                      \
}
/////////////////////////////////////////////////////////////////////////////////////////////////////
// Fill  random numbers on GPU
double fill_rand() {
  double s = 0, d = 0;
  s = rand() % 40 - 20;
  d = s / 137;
  return d;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////
//Print matrix A(rows_A, cols_A) storage in column-major format
void print_matrix( double * A, int rows_A, int cols_A) {


  for (int i = 0; i < rows_A; ++i) {
    for (int j = 0; j < cols_A; ++j) {
      printf("%.2lf ", A[i *cols_A + j]);
      
    }
    printf("\n");

   
  }
printf("\n");
 

}
/////////////////////////////////////////////////////////////////////////////////////////////////////

void calculateElapsedTime(int iterations, float time_elapsed){
       
        cout << "Average time elapsed: " << time_elapsed/iterations << " seconds" << endl;
}
/////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char * * argv) {

  	//fixed srand for debugging
	srand(time(0));

	//select device
	hipSetDevice(0);
	cudaCheckError();

	if (argc < 4)
	{
		cout << "Usage: " << argv[0] << "[Rows]-[Cols]-[Iterations]" << endl;
		return 1;
	}

	unsigned int rows_A, cols_A, rows_C, cols_C, rows_A_T, cols_A_T;

	rows_A = cols_A_T = atoi(argv[1]);
	cols_A = rows_A_T = atoi(argv[2]);
	rows_C = cols_C = cols_A;
	int iterations = atoi(argv[3]);

	hipEvent_t start, stop;

	// Allocate host_A and host_C on host
	double *host_A = (double*) malloc(rows_A *cols_A* sizeof(double));
	double *host_C = (double*) malloc(rows_C *cols_C* sizeof(double));

	//fill array host_A with values in column major order

	double random_data = 1.0;
	for (int i = 0; i < rows_A; ++i)
	{
		for (int j = 0; j < cols_A; ++j)
		{
			host_A[j *rows_A + i] = random_data;
			random_data += 1.0;
		}
	}

	// Allocate array device_A and deive_C on GPU memory
	double *device_A;
	hipMalloc(&device_A, rows_A *cols_A* sizeof(double));
	cudaCheckError();

	double *device_C;
	hipMalloc(&device_C, rows_C *cols_C* sizeof(double));
	cudaCheckError();

	//Copy array host_A to gpu thought device_A
	hipMemcpy(device_A, host_A, rows_A *cols_A* sizeof(double), hipMemcpyHostToDevice);
	cudaCheckError();

	//create cuda events for time measuring
	hipEventCreate(&start);
	cudaCheckError();
	hipEventCreate(&stop);
	cudaCheckError();

	float time_elapsed = 0;
	float tempTime;

	cout << "Executing operation for matrix A with " <<
		cols_A << " columns and " << rows_A << " rows." << endl;
	cout << "=======================================" << endl;

	for (int i = 0; i < iterations; i++)
	{

		tempTime = 0;

	
		double alpha = 1;
		double beta = 0;

		// Create cublas handle
		hipblasHandle_t handle;
		cudaCheckError();
		hipblasCreate(&handle);
		cudaCheckError();
		hipEventRecord(start);
		cudaCheckError();

		hipblasDgemm(handle, HIPBLAS_OP_T,
			HIPBLAS_OP_N, cols_A,
			cols_A, rows_A, &alpha, device_A,
			rows_A, device_A,
			rows_A, &beta,
			device_C, cols_A);
		cudaCheckError();

		hipEventRecord(stop);
		cudaCheckError();

		// Destroy the handle
		hipblasDestroy(handle);
		cudaCheckError();

		// Copy the resulted matrix device_C to host
		hipMemcpy(host_C, device_C, rows_C *cols_C* sizeof(double), hipMemcpyDeviceToHost);
		cudaCheckError();

		hipEventSynchronize(stop);
		cudaCheckError();

		// Compute the elapsed time between the two events start and stop
		hipEventElapsedTime(&tempTime, start, stop);
		cudaCheckError();

		cout << "Iteration " << i + 1 << ": " << tempTime << " seconds" << endl;
		time_elapsed += tempTime;
	}

	print_matrix(host_C, rows_C, cols_C);

	hipEventDestroy(start);
	cudaCheckError();

	hipEventDestroy(stop);
	cudaCheckError();

	cout << "=======================================" << endl;
	//calculate elapsed time
	calculateElapsedTime(iterations, time_elapsed);

	//Free memory on host and device
	hipFree(device_A);
	cudaCheckError();
	free(host_A);

	hipFree(device_C);
	cudaCheckError();
	free(host_C);

	return 0;

}